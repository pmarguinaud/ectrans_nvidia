// AlltoAll for NVLINK-connected GPUs ithin a single server, using CUDA IPC
// All pairs of (distinct) GPUs must return 1 for canAccessPeer
// Alan Gray, NVIDIA


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>

#include <unistd.h>
#include <sched.h>
#include <sys/mman.h>
#include <sys/wait.h>
#include <linux/version.h>


// maximum number of devices supported
#define MAX_DEVICES          32

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

// data structure required for IPC setup 

typedef struct ipcDevices_st
{
  int count;
  int ordinals[MAX_DEVICES];
} ipcDevices_t;



// we need a seperate CUDA stream for each target GPU
static hipStream_t streams[MAX_DEVICES];

// structure to contain pointers to remote array data, and offsets into each for destinatin data.
// we maintain 2 copies (first array dimension), corresponding to MTOL and LTOM trans comms.
// This allows us to only perform setup steps the first time, and re-use all following times. 
static double* outputptrall[2][MAX_DEVICES];
static int roff_remote[2][MAX_DEVICES];


// Initialize IP communicatins
extern "C" void initIPC(double* output_d,int* roff, int mtol_or_ltom){

}


static bool already_initialized[2]={0,0};

static bool notFullPeerAccess=0;

// main externally visible routine for performing AlltoAll comms
extern "C" int Alltoallv_CUDAIPC(double* input, int* len, int* soff, 
				  double* output, int* roff,int mtol_or_ltom){


  return 0; 
  
  
}


